#include "hip/hip_runtime.h"
/* recursion_cut_bth.cu */

/*
 * Copyright (C) 2008-2024 ABINIT Group (MMancini)
 *
 * This file is part of the ABINIT software package. For license information,
 * please see the COPYING file in the top-level directory of the ABINIT source
 * distribution.
 *
 */

#include "cuda_common.h"
#include "cuda_header.h"
#include "cuda_rec_head.h"
#include "cuda_api_error_check.h"
//#include "rec_kernels.cu"

/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/
/* This CUDA module contains the functions (kernels) to perform Recursion
   Method DFT on GPU devices when recrcut!=0.

   Kernels function:
  
   Host function:
   recursion_cut_bth
*/
/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/

void 
recursion_cut_bth(
		  const int trotter, 
		  const int gratio, 
		  const int npt, 
		  const int nrec,           //- Max number of recursion
		  const int nptrec,         //- Max number of points at the same time (depends on GPU)
		  int* max_rec,             //- Out-max recursion to have required precision
		  const cureal beta,
		  const cureal fermie,
		  const cureal tolrec,      //- Tollerance of recursion
		  const cureal inf_vol,     //- Infinitesimal volume
		  const int3* pt0,          //- Coord of initial point
		  const int3* pt1,	    //- Coord of final point
		  const int* ngfft,         //- Total spatial grid
		  int* ngfftrec,            //- Linear sizes of the grid
		  const cureal* T_p,        //- The green kernel
		  const cureal* pot,        //- Potential
		  cureal* an,cureal* bn2)   //- Rec coefficients an and bn2

/* NOTES:
   -The vectors for which to calculate the recursion are set in a
   matrix of size (pth_size X height_max).
   -pth_size= is calculated starting of nfftrec (pth_size>=nfftrec) to make
   GPU memory copy efficient.
   -height_max= is the max number of vector(of size nfftrec) which can be
   putted in the matrix  to make the recursion and it depends on the device.
   -At any call, the recursion is computed for npt points, by a single allocation on the device. 
   -The calculation is made only on min(height_max,pos[1]-pos[0])
   points where pos is the current number of point to compute.
   -oldtonew, for any step in recursion, for any point, the un,unold
   inthe next step are obtaind.
*/
	        
{
 /*------------------------------------------------------------------------------------*/
 /*---------------------------------INITIALIZATION-------------------------------------*/
 /*------------------------------------------------------------------------------------*/
 
 /*-------------- Time setting -----------*/
 float* timing = (float*)calloc(DEBUGLEN,sizeof(float));
 hipEvent_t start; CHECK_CUDA_ERROR( hipEventCreate(&start) );
 hipEvent_t stop;  CHECK_CUDA_ERROR( hipEventCreate(&stop) );

 starttime(&start);//-Start timing: memory allocation+setting

 /*--------------- Sizes of Vectors ---------------*/
 int nfftrec = ngfftrec[0]*ngfftrec[1]*ngfftrec[2];         //- Size of the 3d-grid
 int nfft = ngfft[0]*ngfft[1]*ngfft[2];         //- Size of the total 3d-grid   
 int target = (ngfftrec[0]>>1)*(1+ngfftrec[1]*(1+ngfftrec[2]));

 size_t height_max = (size_t) nptrec;
 size_t totallarg  = (size_t)nfft*sizeof(cureal);    //- Size of real vectors
 size_t largeur  = (size_t)nfftrec*sizeof(cureal);   //- Size of real vectors
 size_t clargeur = (size_t)nfftrec*sizeof(cucmplx); //- Size of Complex vectors
 size_t un_pitch = largeur;  //- Pitch to put multi-vectors in a matrix: intial guess
  
 /*------------------ Grids and Blocks ---------------------------*/
 //-Multi vector grid
 dim3 block(320, 1);
 dim3 grid(((size_t)nfftrec+block.x-1)/block.x,(height_max+block.y-1)/block.y);

 //-Grid to select potential
 dim3 block_pot(16,16);
 dim3 grid_pot(((size_t)ngfftrec[0]+block_pot.x-1)/block_pot.x,((size_t)ngfftrec[1]+block_pot.y-1)/block_pot.y);

 /*--------------------- FFT Planes ------------------------------*/
 hipfftHandle plan_dir;
 /*----------------- FFT of the Green Kernel ---------------------*/ 
 //-Get the green kernel  from host 
 cureal *T_p_gpu = NULL;
 CHECK_CUDA_ERROR( hipMalloc((void**)&T_p_gpu,largeur) );
 CHECK_CUDA_ERROR( hipMemcpy(T_p_gpu,T_p,largeur,hipMemcpyHostToDevice) );
 //-Compute the FFT 
 cucmplx *ZT_p_gpu = NULL;
 CHECK_CUDA_ERROR( hipMalloc((void**)&ZT_p_gpu,clargeur) );
 /*Obtain the FFT of the Green Kernel on device */ 
 realtocmplx <<< ((size_t)nfftrec+320-1)/320,320 >>>(T_p_gpu, ZT_p_gpu,nfftrec);
 CUDA_KERNEL_CHECK("realtocmplx");
 CHECK_CUDA_ERROR( hipFree(T_p_gpu) );
 CHECK_CUDA_ERROR( hipfftPlan3d(&plan_dir,ngfftrec[0],ngfftrec[1],ngfftrec[2],FFT_C2C) );	  
 CHECK_CUDA_ERROR( FFTEXECC2C(plan_dir,ZT_p_gpu ,ZT_p_gpu,HIPFFT_FORWARD) );
 CHECK_CUDA_ERROR( hipfftDestroy(plan_dir) );

 /*------------- Allocation of Matrices on Device ----------------*/
 cureal* vn_gpu    = NULL;
 CHECK_CUDA_ERROR( hipMallocPitch((void**) &vn_gpu,&un_pitch,largeur,height_max) );  

 cureal* un_gpu    = NULL;
 CHECK_CUDA_ERROR( hipMallocPitch((void**) &un_gpu,&un_pitch,largeur,height_max) );

 cureal* unold_gpu = NULL;
 CHECK_CUDA_ERROR( hipMallocPitch((void**) &unold_gpu,&un_pitch,largeur,height_max) );

 cureal* an_gpu    = NULL;
 CHECK_CUDA_ERROR( hipMalloc((void**)&an_gpu,height_max*sizeof(cureal)) );

 cureal* bn2_gpu   = NULL;
 CHECK_CUDA_ERROR( hipMalloc((void**)&bn2_gpu,height_max*sizeof(cureal)) );

 /*----------------- Get the Potential from Host -----------------*/
 cureal* pot_gpu   = NULL;
 CHECK_CUDA_ERROR( hipMalloc((void**)&pot_gpu,totallarg) );
 CHECK_CUDA_ERROR( hipMemcpy(pot_gpu,pot,totallarg,hipMemcpyHostToDevice) );

 /*----------------- Local Potential -----------------*/
 cureal* locpot_gpu   = NULL;
 CHECK_CUDA_ERROR( hipMallocPitch((void**) &locpot_gpu,&un_pitch,largeur,height_max) );
  
 /*------------ Local coordinates of points to calculate ---------------------*/
 int delta = pt0->x+ngfft[0]*(pt0->y+pt0->z*ngfft[1]); //-(virtual) linear initial point
 int final = pt1->x+ngfft[0]*(pt1->y+pt1->z*ngfft[1]); //-(virtual) linear final point
 int pth_size = (int)(un_pitch/sizeof(cureal));  //-Pitched size of vectors 
 int ntranche = (final-delta)+1;//-How many (virtual) pts to compute

 /*------------ Auxiliary Complex Vector -----------*/
 cucmplx* cvn_gpu  = NULL;
 CHECK_CUDA_ERROR( hipMallocPitch((void**) &cvn_gpu,&un_pitch,clargeur,height_max) );
 int cvpthsz = int(un_pitch/sizeof(cucmplx)); //-Pitched size of complex auxiliar vec.
 printf("batched CUFFT -- CUFFT PITCHED SIZES %d %d\n",pth_size,cvpthsz);

 /*------------ Copy Variable in cnst Memory -------------------*/
 copytoconstmem(nfftrec, nptrec, pth_size, cvpthsz);


 /*-------- Initialization points to compute in the First Loop ---*/
 int pos0 = 0;
 int loctranc = min(nptrec,npt);
 int pos1 = pos0+loctranc; 
 *max_rec = 0;

 //-Trotter
 int loctrott = 4*trotter;
 if(trotter == 0) loctrott = 2;

 /*-------- Arrays for Exit Criteria (Density Calculation) ---*/
 cucmplx* ND      = (cucmplx*) malloc(npt*loctrott*sizeof(cucmplx));
 cucmplx* NDold   = (cucmplx*) malloc(npt*loctrott*sizeof(cucmplx));
 cucmplx* NDnew   = (cucmplx*) malloc(npt*loctrott*sizeof(cucmplx));
 cucmplx* acc_rho = (cucmplx*) malloc(npt*sizeof(cucmplx));
 cureal*  erreur  = (cureal*)  malloc(2*npt*sizeof(cureal));
 cureal*  prod_b2 = (cureal*)  malloc(npt*sizeof(cureal));

 calctime(&stop,start,timing,0); //-End Timing: memory allocation+setting

 /*---------------------------------------------------------------------------------*/
 /*---------------- MAIN LOOP on pos1>pos0 -----------------------------------------*/
 /*---------------------------------------------------------------------------------*/
 printf(" Start  %10d\n End    %10d\n Npt    %10d\n gratio %10d\n Tot pt %10d\n",delta,final,npt,gratio,ntranche);

 int ipt;
 int npoint = delta;

 do{starttime(&start);
   int contrec = 0;
   printf("now: from %d to %d, so %d pts of %d \n",pos0+delta,pos1+delta,loctranc,npt);    
   int3 trasl;
   ipt = 0;
   for(int kk=pt0->z;kk<=pt1->z;kk+=gratio){
     trasl.z=kk-(ngfftrec[2]>>1);
     for(int jj=0;jj<ngfft[1];jj+=gratio){
       trasl.y=jj-(ngfftrec[1]>>1);
       for(int ii=0;ii<ngfft[0];ii+=gratio){
	 trasl.x=ii-(ngfftrec[0]>>1);
	 int ipoint = ii+ngfft[0]*(jj+kk*ngfft[1]);
	 if(ipoint<npoint) continue;
	 get_loc_potent <<< grid_pot,block_pot >>> (pot_gpu, locpot_gpu, trasl,ipt, ngfftrec[0],ngfft[0]);
	 CUDA_KERNEL_CHECK("get_loc_potent");
	 //prt_dbg_arr(&(locpot_gpu[ipt*pth_size]),largeur,6,target-3,"potloc");
	 ipt++;
	 if(ipt==loctranc || ipoint == final) {npoint = ipoint+1; goto end_3_loop;}
       }}}
   end_3_loop:  calctime(&stop,start,timing,1);
   check_err(0);
    
   /*--------- Setting arrays Un,Unold on the Device --------------*/  
   starttime(&start);
   setting_un_cut <<< grid, block >>>(un_gpu,unold_gpu,vn_gpu,an_gpu, bn2_gpu,rsqrt(inf_vol),target);
   CUDA_KERNEL_CHECK("setting_un_cut");
   calctime(&stop,start,timing,1);
   //prt_dbg_arr(un_gpu,largeur,10,0,"un0");
   check_err(0);
  
   CHECK_CUDA_ERROR( hipfftPlanMany(&plan_dir,3,ngfftrec,NULL,1,0,NULL,1,0,FFT_C2C,loctranc) );

   /*------------------ Loop on nrec ------------------------------*/
   int irec;
   for(irec=0; irec<nrec+1; irec++){
#ifdef  HAVE_GPU_CUDA_DEBUG
     printf("IREC------------%d\n",irec);
#endif
     starttime(&start);
     //prt_dbg_arr(un_gpu,largeur,6,target-3+pth_size,"un_gpu");
     un_x_pot_cut <<< grid, block >>> (cvn_gpu,un_gpu, locpot_gpu, loctranc);
     CUDA_KERNEL_CHECK("un_x_pot_cut");
     calctime(&stop,start,timing,2);
     //prt_dbg_arr(vn_gpu,largeur,6,target-3+pth_size,"vn=un*pot");

     /*-------------- Loop on loctranc: CONVOLUTION by FFT -----*/
      
     /*----- FFT -----*/
     starttime(&start);
     CHECK_CUDA_ERROR( FFTEXECC2C(plan_dir,cvn_gpu,cvn_gpu,HIPFFT_FORWARD) );
     calctime(&stop,start,timing,3);

     /*---- Moltiplication of the FFT with the Green Kernel ------*/
     starttime(&start); 
     complex_prod_tot <<< grid, block >>> (cvn_gpu, ZT_p_gpu,loctranc,nfftrec);
     CUDA_KERNEL_CHECK("complex_prod_tot");
     calctime(&stop,start,timing,4);

     /*---- Inverse FFT -----*/
     starttime(&start); 
     CHECK_CUDA_ERROR( FFTEXECC2C(plan_dir,cvn_gpu,cvn_gpu,HIPFFT_BACKWARD) );
     calctime(&stop,start,timing,3);

       
     /*-------------- Compute Vn = Pot*Vn -------------*/
     starttime(&start);
     vn_x_pot_dv_cut <<< grid, block >>>(cvn_gpu,vn_gpu, locpot_gpu,inf_vol/cureal(nfftrec), loctranc);
     CUDA_KERNEL_CHECK("vn_x_pot_dv_cut");
     calctime(&stop,start,timing,5);   
     //prt_dbg_arr(vn_gpu,largeur,6,target-3,"vn=vn*pot*infvol2");

     /*-------------- Compute An = Un*Vn -------------*/
     starttime(&start);
     scalarProdGPU<<< 128,256 >>>(an_gpu, vn_gpu, un_gpu, inf_vol);
     CUDA_KERNEL_CHECK("scalarProdGPU");
     //-Copying An on the host
     CHECK_CUDA_ERROR( hipMemcpy(&(an[(irec)*npt+pos0]),an_gpu,(size_t)loctranc*sizeof(cureal),hipMemcpyDeviceToHost) );
     calctime(&stop,start,timing,6);
     //prt_dbg_arr(an_gpu,(height_max)*sizeof(cureal),10,0,"an");
   
     /*-------------- PREPARING NEXT ITERATION IN IREC -----------------*/
     if(irec<nrec){
       /*---------- Compute Un,Vn,Unold: Old to New -------*/
       starttime(&start);
       oldtonew <<< grid,block >>> (un_gpu,vn_gpu,unold_gpu,an_gpu,bn2_gpu,loctranc);
       CUDA_KERNEL_CHECK("oldtonew");
       calctime(&stop,start,timing,7);

       /*---------- Compute Bn = Un*Un ---------------*/
       starttime(&start);
       scalarProdGPU <<< 128,256 >>> (bn2_gpu, un_gpu, un_gpu, inf_vol);
       CUDA_KERNEL_CHECK("scalarProdGPU");
       //-Copying An on the host
       CHECK_CUDA_ERROR( hipMemcpy(&(bn2[(irec+1)*npt+pos0]),bn2_gpu,loctranc*sizeof(cureal),hipMemcpyDeviceToHost) );
       calctime(&stop,start,timing,8);
       //prt_dbg_arr(bn2_gpu,(height_max)*sizeof(cureal),10,0,"bn2");
	
       /*---------- Compute Un = Un/Sqrt(Bn) ---------------*/
       starttime(&start);
       un_invsqrt_scale <<< grid,block >>> (un_gpu, bn2_gpu, loctranc);
       CUDA_KERNEL_CHECK("un_invsqrt_scale");
       calctime(&stop,start,timing,9);
       //prt_dbg_arr(un_gpu,largeur,6,target-3+pth_size,"unnew rescaled");

 
       /*--------- Exit Criterium: Density and Error Calculations ----------*/
       starttime(&start);
       density_calc( beta*fermie, 2./inf_vol,tolrec,irec,trotter,npt,loctranc,
		     pos0,&(contrec), bn2, an,
		     erreur,prod_b2, 
		     acc_rho, ND, NDold, NDnew);
       calctime(&stop,start,timing,10);
       if(contrec==loctranc) break;
     }     
   }//End loop on nrec

   *max_rec = max(*max_rec,irec+1);     

   CHECK_CUDA_ERROR( hipfftDestroy(plan_dir) );
    
   /*-------- Points to compute in the Next Loop ---*/
   loctranc = min(nptrec,npt-pos1);
   pos0 = pos1; 
   pos1 = min(pos0+loctranc,npt-1);
   
 }while(pos0 <pos1);

#if defined HAVE_GPU_CUDA3
 CHECK_CUDA_ERROR( hipDeviceSynchronize() );
#else
 CHECK_CUDA_ERROR( hipDeviceSynchronize() );
#endif   

 /*--------------Free Memory on Device and Host----------*/
 free(ND);
 free(NDold);
 free(NDnew);
 free(acc_rho);
 free(erreur);
 free(prod_b2);

 CHECK_CUDA_ERROR( hipFree(cvn_gpu) );
 CHECK_CUDA_ERROR( hipFree(ZT_p_gpu) );
 CHECK_CUDA_ERROR( hipFree(pot_gpu) );
 CHECK_CUDA_ERROR( hipFree(locpot_gpu) );
 CHECK_CUDA_ERROR( hipFree(vn_gpu) );
 CHECK_CUDA_ERROR( hipFree(unold_gpu) );
 CHECK_CUDA_ERROR( hipFree(un_gpu) );
 CHECK_CUDA_ERROR( hipFree(an_gpu) );
 CHECK_CUDA_ERROR( hipFree(bn2_gpu) );

 /*____________________Printing Time_____________________*/
 prt_device_timing(timing,DEBUGLEN);
 free(timing);
 hipEventDestroy(start);
 hipEventDestroy(stop);
 
 printf("\n--end--cudarec cut------ \n"); 
 return;
}

