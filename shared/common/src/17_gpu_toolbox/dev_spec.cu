#include "hip/hip_runtime.h"
/* dev_spec.cu*/

/*
 * Copyright (C) 2008-2022 ABINIT Group (MMancini,FDahm)
 * this file is distributed under the terms of the
 * gnu general public license, see ~abinit/COPYING
 * or http://www.gnu.org/copyleft/gpl.txt.
 *
 */

#include <stdio.h>
#include <abi_gpu_header_common.h>

static __host__ int version_2_cores(int major, int minor);

/*=========================================================================*/
/*________________________ GPU_function called by HOST_____________________*/
/*=========================================================================*/
// display CUDA device info
static __host__ void  prt_dev_info()
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  for (int dev = 0; dev < deviceCount; ++dev)
    {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, dev);
      int NProcs=deviceProp.multiProcessorCount;
      int NCores=version_2_cores(deviceProp.major, deviceProp.minor);
      printf("\n___________________________________________________________________\n");
      printf(  "__________  Graphic Card Properties  ______________________________\n");
      printf("\n  Device %d: \"%s\"\n", dev, deviceProp.name);
      printf("  Revision number:                               %d.%d\n", deviceProp.major,deviceProp.minor);
      printf("  Total amount of global memory:                 %3.1f Mbytes\n", deviceProp.totalGlobalMem/1048576.);
      printf("  Clock rate:                                    %3.1f GHz\n", deviceProp.clockRate/1000000.);
      printf("  Number of processors/cores:                    %d/%d\n", NProcs,NCores);
      if (NCores<0) {
        printf("  Max GFLOPS:                                    undefined (add new def. in version_2_cores function)\n");
      } else {
        printf("  Max GFLOPS:                                    %d GFP\n", NCores*deviceProp.multiProcessorCount * deviceProp.clockRate/1000000);
      }
      printf("  Total amount of constant memory:               %d bytes\n",(int) deviceProp.totalConstMem);
      printf("  Total amount of shared memory per block:       %d bytes\n",(int) deviceProp.sharedMemPerBlock);
      printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
      printf("___________________________________________________________________\n");
      fflush(stdout);
      if( (int) deviceProp.totalConstMem<0) break;
      //if(deviceProp.major==9999){printf("EXIT: PROBLEM WITH AVAILABLE DEVICES \n");exit(0);}
    }
}


// Explicit Cuda Error ---------------------
__host__  void
check_err(int line ){
/* cuda check errors */
  hipError_t hipError_t;
  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
    { fprintf(stderr, "CUDA Runtime API Error reported : %s %d\n", hipGetErrorString(hipError_t),line);
      exit(EXIT_FAILURE);
    }
  return;
}


// Gives the number of GPU devices ---------
extern "C" __host__
void get_gpu_ndev_(int* ndevice)
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  *ndevice = deviceCount;

  return;
}

// Gives the max memory available for a GPU device ---------
extern "C" __host__
void get_gpu_max_mem_(int* device, float* max_mem)
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, *device);
  *max_mem = (float) deviceProp.totalGlobalMem;
   return;
}


// Set the device if it exists   -----------------
extern "C" __host__
void set_dev_(int* gpudevice)
{
 if(*gpudevice >-1){
   hipError_t hipError_t;
   int deviceCount;
   hipGetDeviceCount(&deviceCount);
   if(deviceCount>*gpudevice){
     hipSetDevice(*gpudevice);
     hipError_t = hipGetLastError();
     if(hipError_t != hipSuccess){
       fprintf(stderr, "CUDA Runtime API Error reported : %s\n", hipGetErrorString(hipError_t));
       fflush(stderr);
       exit(1);
     }
   }
   else *gpudevice=-1;
 }
  return;
}


// Unset the devices  -----------------
extern "C"  __host__
void unset_dev_()
{
#if defined HAVE_GPU_CUDA3
  hipDeviceReset();
#else
  hipDeviceReset();
#endif
  return;
}


// Get context  -----------------------
extern "C"  __host__
void check_context_(int *res,char *message)
{
  *res=1;
  hipError_t state=hipFree(0);
  if (state!=hipSuccess){
    sprintf(message,"Unable to initialize a Cuda context: %s \n",hipGetErrorString(state));
    *res=0;
    unset_dev_();
  }
}


// Get info from device  --------------
extern "C" __host__
void  get_dev_info_(int* device,
		    char* name,
		    int* lenname,
		    int vers[2],
		    float* globalmem,
		    float* clockrate,
		    int* gflops,
		    int* constmem,
		    int* sharemem,
		    int* regist,
		    int* nprocs,
		    int* ncores
		    )
{
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, *device);
  strcpy(name,deviceProp.name);
  *lenname = strlen( name );
  vers[0] = deviceProp.major;
  vers[1] = deviceProp.minor;
  *globalmem = deviceProp.totalGlobalMem/1048576.;
  *clockrate = deviceProp.clockRate/1000000.;
  *nprocs = deviceProp.multiProcessorCount;
  *ncores = version_2_cores(deviceProp.major,deviceProp.minor);
  *gflops = int(deviceProp.multiProcessorCount*version_2_cores(deviceProp.major,deviceProp.minor)*(deviceProp.clockRate/1000000.));
  *constmem = deviceProp.totalConstMem;
  *sharemem =  deviceProp.sharedMemPerBlock;
  *regist = deviceProp.regsPerBlock;
}


// Get number of devices  --------------
extern "C"  __host__
void c_get_ndevice_(int* ndev)
{
  *ndev=0;
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  for (int idev = 0; idev < deviceCount; ++idev)
    {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, idev);
      //We check that no device is in "emu" mode
      if( deviceProp.major != 9999 ) {
#if defined HAVE_GPU_CUDA_DP
      //We check that double precision is available, c.c. >= 1.3 )
      if( (deviceProp.major>1)||(deviceProp.minor>2) )
#endif
	*ndev+=1;
      }
    }
}


// Get number of cores of device  --------------
//This function is present in cuda SDK: see ${CUDAROOT}/common/inc/helper_cuda_drvapi.h
//To be completed for new card versions
static __host__
int version_2_cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;
    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x32, 192}, // Kepler Generation (SM 3.2) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        { 0x50, 128}, // Maxwell Generation (SM 5.0) GM10x class
        { 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
        { 0x61, 128}, // Pascal Generation (SM 6.1) GP10x class
        { 0x62, 128}, // Pascal Generation (SM 6.2) GP10x class
        { 0x70, 64 }, // Volta Generation (SM 7.0) GV100 class
        { 0x72, 64 }, // Volta Generation (SM 7.2) AGX class
        { 0x75, 64 }, // Turing Generation (SM 7.5) RTX class
        { 0x80, 64 }, // Ampere Generation (SM 8.0) A100 class
        { 0x86, 128}, // Ampere Generation (SM 8.6)
        { 0x87, 128}, // Ampere Generation (SM 8.7)
        {   -1, -1 }
    };
    int index = 0;
    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }
        index++;
    }

//  printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[7].Cores);
    return nGpuArchCoresPerSM[10].Cores;
}


/***************************************************************/
/*******                                                ********/
/*******      GPU MEMORY MANAGEMENT ROUTINES            ********/
/*******                                                ********/
/***************************************************************/

/*============================================================================*/
/* Print memory information (total amount and free available)                 */
/*============================================================================*/

extern "C" void check_gpu_mem_(){
  size_t free,total;
  hipMemGetInfo(&free,&total);
  printf("*** GPU memory : Free =>  %4.2fMo   | Total =>  %4.2fMo ***\n",free*1e-6,total*1e-6);
  fflush(stdout);
}

/*============================================================================*/
/* Allocate size byte in gpu memory and returns in gpu_ptr this location      */
/* INPUTS size= size in byte to allocate                                      */
/* OUTPUT gpu_ptr= C_PTR on gpu memory location that has been allocated       */
/*============================================================================*/

extern "C" void alloc_on_gpu_(void **gpu_ptr,int* size){

  if(hipMalloc(gpu_ptr,*size)!=hipSuccess){
    printf("ERROR: alloc_on_gpu failed:%s\n",hipGetErrorString(hipGetLastError()));
    fflush(stdout);
    abi_cabort();
  }
}

/*============================================================================*/
/* Free memory location pointed by gpu_ptr                                    */
/* OUTPUT gpu_ptr= C_PTR on gpu memory location that has been allocated       */
/* WARNING! : this routine is a dummy one when HAVE_GPU_CUDA is not enabled   */
/*            the correct one is in xx_gpu_toolbox/dev_spec.cu                */
/*============================================================================*/

extern "C" void dealloc_on_gpu_(void **gpu_ptr){
  if(*gpu_ptr==NULL)
    return;
  if(hipFree(*gpu_ptr)!=hipSuccess){
    printf("ERROR: dealloc_on_gpu failed :%s\n",hipGetErrorString(hipGetLastError()));
    fflush(stdout);
    abi_cabort();
  }
  *gpu_ptr=NULL;
}

/*============================================================================*/
/* Copy size byte from  dtab to gpu memory pointed by gpu_ptr                 */
/* INPUTS                                                                     */
/*  size= size in byte to allocate                                            */
/*  dtab = fortran tab to copy                                                */
/* OUTPUT                                                                     */
/*  gpu_ptr= C_PTR on gpu memory location                                     */
/* WARNING! : this routine is a dummy one when HAVE_GPU_CUDA is not enabled   */
/*            the correct one is in xx_gpu_toolbox/dev_spec.cu                */
/*============================================================================*/

extern "C" void copy_on_gpu_(void *ptr, void **gpu_ptr,int* size){
  if(hipMemcpy(*gpu_ptr,ptr,*size,hipMemcpyHostToDevice)!=hipSuccess){
    printf("ERROR: copy_on_gpu failed : %s\n",hipGetErrorString(hipGetLastError()));
    fflush(stdout);
    abi_cabort();
  }
}

/*============================================================================*/
/* Copy size byte from gpu memory pointed by gpu_ptr to dtab                  */
/* INPUTS                                                                     */
/*  size= size in byte to allocate                                            */
/*  gpu_ptr= C_PTR on gpu memory location that has been allocated             */
/* OUTPUT                                                                     */
/*  dtab = fortran tab which will contains data                               */
/*============================================================================*/

extern "C" void copy_from_gpu_(void *ptr,void **gpu_ptr,int* size){
  if(hipMemcpy(ptr,*gpu_ptr,*size,hipMemcpyDeviceToHost)!=hipSuccess){
    printf("ERROR: copy_from_gpu failed : %s\n",hipGetErrorString(hipGetLastError()));
    fflush(stdout);
    abi_cabort();
  }
}
